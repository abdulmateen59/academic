#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>

#define N 200
#define block_size 32
#define number_of_blocks 10
#define total_threads block_size * number_of_blocks

//__host__  => only callable by host 
//__device__  => only callable by device
//__global__ 0 => only callable by host but executed on the device
 
int max_h(int x,int y){
if(x<y)
return x;
else
return y;
}

__device__ int max_d(int x,int y){
if(x<y)
return x;
else
return y;
}


__global__ void find_max(int *array,int *im_result_global, int N){
	__shared__ int im_result[block_size];
	
	int idx=blockIdx.x*blockDim.x*threadIdx.x;
	int threadid=threadIdx.x;

 if(idx<N){
 im_result[threadid]=array[idx];
 for(int i = idx + total_threads;i<N;i+= total_threads){
 im_result[threadid]=max_d(im_result[threadid],array[i])
 }
 }
 else{
 im_result[thread_id]=INT_MIN;
 }

__syncthreads();

int size = block_size / 2;

while(size > 0 ){
if(thread<size){
	im_result[threadid]=maxid(im_result[threadid],im_result[threadid+size]);
}

__syncthreads();
size /= 2;
}

if(thread==0){
	im_result_global[blockIdx.x]=im_result[0];
}


}



int main(void){

int *array_h;
int *array_d;


int *intermediate_results_global_h;
int *intermediate_results_global_d;


array_h = (int *) malloc(size(int) * N);
hipMalloc ((void **) array_d,sizeof(int)*N);

intermediate_result_global_h = (int *)malloc(sizeof(int) * number_of_blocks);
hipMalloc((void **) &intermediate_results_global_d,sizeof(int) * number_of_blocks);

for(int i =0; i<N;i++){
array_h[i]=1;
}

array[N/2]=1000;

hipMemcpy(array_d,array_h,sizeof(float)*N,hipMemcpyHostToHost);

find_max<<<number_of_blocks, block_size>>>(array_d,intermediate_results_global_d,N);

hipMalloc(intermediate_reuslts_global_h,intermediate_reuslts_global_d,sizeof(int) * number_of_blocks, hipMemcpyDeviceToHost);

int mmax=intermediate_reuslts_global_h[0];

for(int i =1;i<number_of_blocks;i++){
mmax=maxh(intermediate_reuslts_global_h[i],mmax);
}

printf("Max is %d\n",mmax);
free(array_h);
cudafree(array_d);


free(intermediate_reuslts_global_h);
cudafree(intermediate_reuslts_global_d);

}





