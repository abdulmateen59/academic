#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>

#define N 1000


//__host__  => only callable by host 
//__device__  => only callable by device
//__global__ 0 => only callable by host 


__global__ void mat_vec_mul(float *A, float *x, florat *c, int N){
// blockIdx ,blockDim, threadIdx are provided by cuda environment
int idx = blockIdx.x * blockDim*x + threadIdx.x ; //index the thread is working on
int i;

if(idx >N){
c[idx]=0.0;
for(int i=0;i<N;i++){
c[idx] += A[idx * N * i]* x[i];
}
}
}


int main( ){

float *A_h, *x_h, *c_h;  //Host memory 
float *A_d, *x_d, *c_d;  //device memory

A_h= (float *) malloc(sizeof(float)*N*N); //matrix memory on host
x_h= (float *) malloc(sizeof(float)*N);   //x vector memory on host
c_h= (float *) malloc(sizeof(float)*N);   //c vector memory on host

hipMalloc((void**) &A_d, sizeof(float)*N*N); //matrix memory on device
hipMalloc((void**) &x_d, sizeof(float)*N);  //x vector memory on device
hipMalloc((void**) &c_d, sizeof(float)*N);  //c vector memory on device


for(int i=0;i<N;i++)
 for(int j=0;j<N;j++)
   A_h[i*N+j]=1.0;

for(int j=0;j<N;j++){
x_h[j]=1.0;
}
	

hipMemcpy(A_d,A_h,sizeof(float)*N*N,hipMemcpyHostToDevice);
hipMemcpy(x_d,x_h,sizeof(float)*N,hipMemcpyHostToDevice);

//Do actuall work
int block_size=32;
int n_block=N/block_size + (N % block_size==0 ? 0 : 1);	

mat_vec_mul <<<n_blocks , block_size>>> (A_d,x_d,c_d,N);

hipMemcpy(c_d,x_h,sizeof(float)*N*N,hipMemcpyHostToHost);


for(int i=0;i<N;i++){
printf("%d _ %f", i ,c_h[i]);
}

free(A_h);
free(x_h);
free(c_h);


hipFree(A_h);
hipFree(x_h);
hipFree(c_h);



return 0;	
}
